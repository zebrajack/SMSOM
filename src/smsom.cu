#include "hip/hip_runtime.h"
/*
 * This single source code is a straightforward implementation of our paper [1]. See http://zhaozj89.github.io/SMSOM/ for more details.
 * Written by Zhenjie Zhao, if you have any question, please feel free to contact <zhaozj89@gmail.com>.
 *
 * The executable smsom.exe in /SMSOM/Debug is built under the following environment:
 * 1. Visual Studio 2010
 * 2. CUDA 5.0
 * 3. OpenCV 2.4.5
 * 4. Windows 7 (64 bit)
 * You can use the executable directly in a similar environment. Alternatively, you can build it in other environments manually. See README.md file for more details.
 *
 * [1] Zhenjie Zhao, Xuebo Zhang, and Yongchun Fang. Stacked Multi-layer Self-Organizing Map for Background Modeling. IEEE Transactions on Image Processing, 2015, Accepted.
 */

// cuda5.0
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

// OpenCV
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"

// std
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <cstring>

//
using namespace cv;
using namespace std;

//
__device__ const float PI = 3.1415926;
__device__ float gaussKernel[3][3] = {1/16.0, 2/16.0, 1/16.0, 2/16.0, 4/16.0, 2/16.0, 1/16.0, 2/16.0, 1/16.0};

__device__ int mi[9] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
__device__ int mj[9] = {0, 0, 0, 1, 1, 1, 2, 2, 2};

__device__ int xlu[9] = {-1, 0, 0, -1, 0, 0, -1, 0, 0};
__device__ int xu[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
__device__ int xru[9] = {0, 0, 1, 0, 0, 1, 0, 0, 1};
__device__ int xr[9] = {0, 0, 1, 0, 0, 1, 0, 0, 1};
__device__ int xrd[9] = {0, 0, 1, 0, 0, 1, 0, 0, 1};
__device__ int xd[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
__device__ int xld[9] = {-1, 0, 0, -1, 0, 0, -1, 0, 0};
__device__ int xl[9] = {-1, 0, 0, -1, 0, 0, -1, 0, 0};

__device__ int ylu[9] = {-1, -1, -1, 0, 0, 0, 0, 0, 0};
__device__ int yu[9] = {-1, -1, -1, 0, 0, 0, 0, 0, 0};
__device__ int yru[9] = {-1, -1, -1, 0, 0, 0, 0, 0, 0};
__device__ int yr[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
__device__ int yrd[9] = {0, 0, 0, 0, 0, 0, 1, 1, 1};
__device__ int yd[9] = {0, 0, 0, 0, 0, 0, 1, 1, 1};
__device__ int yld[9] = {0, 0, 0, 0, 0, 0, 1, 1, 1};
__device__ int yl[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ int ilu[9] = {2, 0, 1, 2, 0, 1, 2, 0, 1};
__device__ int iu[9] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
__device__ int iru[9] = {1, 2, 0, 1, 2, 0, 1, 2, 0};
__device__ int ir[9] = {1, 2, 0, 1, 2, 0, 1, 2, 0};
__device__ int ird[9] = {1, 2, 0, 1, 2, 0, 1, 2, 0};
__device__ int id[9] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
__device__ int ild[9] = {2, 0, 1, 2, 0, 1, 2, 0, 1};
__device__ int il[9] = {2, 0, 1, 2, 0, 1, 2, 0, 1};

__device__ int jlu[9] = {2, 2, 2, 0, 0, 0, 1, 1, 1};
__device__ int ju[9] = {2, 2, 2, 0, 0, 0, 1, 1, 1};
__device__ int jru[9] = {2, 2, 2, 0, 0, 0, 1, 1, 1};
__device__ int jr[9] = {0, 0, 0, 1, 1, 1, 2, 2, 2};
__device__ int jrd[9] = {1, 1, 1, 2, 2, 2, 0, 0, 0};
__device__ int jd[9] = {1, 1, 1, 2, 2, 2, 0, 0, 0};
__device__ int jld[9] = {1, 1, 1, 2, 2, 2, 0, 0, 0};
__device__ int jl[9] = {0, 0, 0, 1, 1, 1, 2, 2, 2};

__device__ bool shadowRemove(float hi, float si, float vi,
																													float hm, float sm, float vm){
								return ( (vi/vm<1) && (vi/vm>0.7) && (si-sm<0.1) && (fabs(hi-hm)<10) );
}

__device__ float distance(float h1, float s1, float v1,
																										float h2, float s2, float v2){
								return sqrtf(pow(s1*v1*cos(h1*PI/180) - s2*v2*cos(h2*PI/180), 2) +
																					pow(s1*v1*sin(h1*PI/180) - s2*v2*sin(h2*PI/180), 2) +
																					pow(v1 - v2, 2));
}

//
__global__ void initLayer(float* input, float* output, int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y; // thread index

								for (int j=0; j<3; ++j) {
																for (int i=0; i<3; ++i) {
																								output[(y*3+j)*width*3+(x*3+i)] = input[y*width+x];
																}
								}
}

//
__global__ void compete(float* modelH, float* modelS, float* modelV,
																								float* frameH, float* frameS, float* frameV,
																								bool* match, int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								pointModel[j*3+i][0] = modelH[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][1] = modelS[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][2] = modelV[(y*3+j)*width*3+(x*3+i)];
																}
								}

								int index = 0;
								int i2 = 0;
								float min = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

								for (int i2 = 1; i2 < 3*3; ++i2) {
																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																if (dist <= min) {
																								min = dist;
																								index = i2;
																}
								}

								for (int j3 = 0; j3 < 3; ++j3) {
																for (int i3 = 0; i3 < 3; ++i3) {
																								match[(y*3+j3)*width*3+(x*3+i3)] = false;
																}
								}
								match[(y*3+mj[index])*width*3+(x*3+mi[index])] = true;
}

__global__ void competeWithFilter(float* model1H, float* model1S, float* model1V,
																																		float* model2H, float* model2S, float* model2V,
																																		float* frameH, float* frameS, float* frameV,
																																		float* maxValue,
																																		bool* match, int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
																}
								}

								int i2 = 0;
								float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

								for (int i2 = 1; i2 < 3*3; ++i2) {
																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																if (dist >= max)
																								max = dist;
								}

								for (int j3 = 0; j3 < 3; ++j3) {
																for (int i3 = 0; i3 < 3; ++i3) {
																								match[(y*3+j3)*width*3+(x*3+i3)] = false;
																}
								}

								if( max >= maxValue[y*width+x] ) {
																for (int j = 0; j < 3; ++j) {
																								for (int i = 0; i < 3; ++i) {
																																pointModel[j*3+i][0] = model2H[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][1] = model2S[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][2] = model2V[(y*3+j)*width*3+(x*3+i)];
																								}
																}

																int index = 0;
																int i2 = 0;
																float min = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																					pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

																for (int i2 = 1; i2 < 3*3; ++i2) {
																								float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																														pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																								if (dist <= min) {
																																min = dist;
																																index = i2;
																								}
																}
																match[(y*3+mj[index])*width*3+(x*3+mi[index])] = true;
								}
}

__global__ void competeWithFilter2(float* model1H, float* model1S, float* model1V,
																																			float* model2H, float* model2S, float* model2V,
																																			float* model3H, float* model3S, float* model3V,
																																			float* frameH, float* frameS, float* frameV,
																																			float* maxValue1,
																																			float* maxValue2,
																																			bool* match, int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
																}
								}

								int i2 = 0;
								float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

								for (int i2 = 1; i2 < 3*3; ++i2) {
																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																if (dist >= max)
																								max = dist;
								}

								for (int j3 = 0; j3 < 3; ++j3) {
																for (int i3 = 0; i3 < 3; ++i3) {
																								match[(y*3+j3)*width*3+(x*3+i3)] = false;
																}
								}

								if( max >= maxValue1[y*width+x] ) {
																for (int j = 0; j < 3; ++j) {
																								for (int i = 0; i < 3; ++i) {
																																pointModel[j*3+i][0] = model2H[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][1] = model2S[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][2] = model2V[(y*3+j)*width*3+(x*3+i)];
																								}
																}

																int i2 = 0;
																float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																					pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

																for (int i2 = 1; i2 < 3*3; ++i2) {
																								float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																														pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																								if (dist >= max)
																																max = dist;
																}

																if( max >=maxValue2[y*width+x] ) {
																								for (int j = 0; j < 3; ++j) {
																																for (int i = 0; i < 3; ++i) {
																																								pointModel[j*3+i][0] = model3H[(y*3+j)*width*3+(x*3+i)];
																																								pointModel[j*3+i][1] = model3S[(y*3+j)*width*3+(x*3+i)];
																																								pointModel[j*3+i][2] = model3V[(y*3+j)*width*3+(x*3+i)];
																																}
																								}

																								int index = 0;
																								int i2 = 0;
																								float min = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

																								for (int i2 = 1; i2 < 3*3; ++i2) {
																																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																																if (dist <= min) {
																																								min = dist;
																																								index = i2;
																																}
																								}
																								match[(y*3+mj[index])*width*3+(x*3+mi[index])] = true;
																} // if
								} // if
}

// update the background model
__global__ void cooperate(float* modelH, float* modelS, float* modelV,
																										float* backupH, float* backupS, float* backupV,
																										float* frameH, float* frameS, float* frameV,
																										bool* match,
																										int width, int height, float alpha){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								int m = 0;
								for(int j = 0; j < 3; ++j) {
																for(int i = 0; i < 3; ++i) {
																								m = j*3+i;
																								// center
																								if(match[(y*3+j)*width*3+(x*3+i)] == true) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][1]*backupH[(y*3+j)*width*3+(x*3+i)]
																																								+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][1]*backupS[(y*3+j)*width*3+(x*3+i)]
																																								+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][1]*backupV[(y*3+j)*width*3+(x*3+i)] +
																																								alpha*gaussKernel[1][1]*(frameV[y*width+x]);
																								}
																								// left up
																								if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 &&
																														match[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][2]*backupH[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																								+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][2]*backupS[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																								+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][2]*backupV[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																								+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
																								}
																								// up
																								if (  (y+yu[m])>=0 &&
																														match[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][1]*backupH[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																								+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][1]*backupS[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																								+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][1]*backupV[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																								+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
																								}
																								// right up
																								if (  (x+xru[m])<=width && (y+yru[m])>=0 &&
																														match[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][0]*backupH[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																								+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][0]*backupS[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																								+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[2][0]*backupV[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																								+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
																								}
																								// right
																								if (  (x+xr[m])<=width &&
																														match[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][0]*backupH[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																								+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][0]*backupS[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																								+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][0]*backupV[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																								+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
																								}
																								// right down
																								if (  (x+xrd[m])<=width && (y+yrd[m])>=height &&
																														match[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][0]*backupH[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																								+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][0]*backupS[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																								+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][0]*backupV[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																								+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
																								}
																								// down
																								if (  (y+yd[m])>=height &&
																														match[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][1]*backupH[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																								+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][1]*backupS[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																								+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][1]*backupV[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																								+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
																								}
																								// left down7
																								if (  (y+yld[m])>=height && (x+xld[m])>=0 &&
																														match[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][2]*backupH[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																								+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][2]*backupS[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																								+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[0][2]*backupV[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																								+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
																								}
																								// left
																								if (  (x+xl[m])>=0 &&
																														match[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ) {
																																modelH[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][2]*backupH[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																								+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
																																modelS[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][2]*backupS[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																								+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
																																modelV[(y*3+j)*width*3+(x*3+i)] =
																																								(1-alpha)*gaussKernel[1][2]*backupV[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																								+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
																								}
																}
								}
}

__global__ void cooperateWithFilter(float* model1H, float* model1S, float* model1V,
																																				float* model2H, float* model2S, float* model2V,
																																				float* backup2H, float* backup2S, float* backup2V,
																																				float* frameH, float* frameS, float* frameV,
																																				float* maxValue,
																																				bool* match,
																																				int width, int height, float alpha){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
																}
								}

								int i2 = 0;
								float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

								for (int i2 = 1; i2 < 3*3; ++i2) {
																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																if (dist >= max)
																								max = dist;
								}

								if( max >= maxValue[y*width+x] ) {
																int m = 0;
																for(int j = 0; j < 3; ++j) {
																								for(int i = 0; i < 3; ++i) {
																																m = j*3+i;
																																// center
																																if(match[(y*3+j)*width*3+(x*3+i)] == true) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][1]*backup2H[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][1]*backup2S[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][1]*backup2V[(y*3+j)*width*3+(x*3+i)] +
																																																alpha*gaussKernel[1][1]*(frameV[y*width+x]);
																																}
																																// left up
																																if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 &&
																																						match[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][2]*backup2H[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																																+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][2]*backup2S[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																																+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][2]*backup2V[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																																+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
																																}
																																// up
																																if (  (y+yu[m])>=0 &&
																																						match[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][1]*backup2H[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																																+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][1]*backup2S[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																																+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][1]*backup2V[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																																+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
																																}
																																// right up
																																if (  (x+xru[m])<=width && (y+yru[m])>=0 &&
																																						match[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][0]*backup2H[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																																+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][0]*backup2S[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																																+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[2][0]*backup2V[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
																																}
																																// right
																																if (  (x+xr[m])<=width &&
																																						match[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][0]*backup2H[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																																+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][0]*backup2S[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																																+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][0]*backup2V[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																																+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
																																}
																																// right down
																																if (  (x+xrd[m])<=width && (y+yrd[m])>=height &&
																																						match[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][0]*backup2H[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																																+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][0]*backup2S[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																																+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][0]*backup2V[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																																+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
																																}
																																// down
																																if (  (y+yd[m])>=height &&
																																						match[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][1]*backup2H[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																																+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][1]*backup2S[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																																+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][1]*backup2V[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																																+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
																																}
																																// left down7
																																if (  (y+yld[m])>=height && (x+xld[m])>=0 &&
																																						match[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][2]*backup2H[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																																+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][2]*backup2S[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																																+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[0][2]*backup2V[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
																																}
																																// left
																																if (  (x+xl[m])>=0 &&
																																						match[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][2]*backup2H[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																																+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][2]*backup2S[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																																+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha)*gaussKernel[1][2]*backup2V[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																																+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
																																}
																								}
																}
								}
}

__global__ void cooperateWithFilter2(float* model1H, float* model1S, float* model1V,
																																					float* model2H, float* model2S, float* model2V,
																																					float* model3H, float* model3S, float* model3V,
																																					float* backup3H, float* backup3S, float* backup3V,
																																					float* frameH, float* frameS, float* frameV,
																																					float* maxValue1,
																																					float* maxValue2,
																																					bool* match,
																																					int width, int height, float alpha){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
																}
								}

								int i2 = 0;
								float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

								for (int i2 = 1; i2 < 3*3; ++i2) {
																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																if (dist >= max)
																								max = dist;
								}

								if( max >= maxValue1[y*width+x] ) {
																for (int j = 0; j < 3; ++j) {
																								for (int i = 0; i < 3; ++i) {
																																pointModel[j*3+i][0] = model2H[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][1] = model2S[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][2] = model2V[(y*3+j)*width*3+(x*3+i)];
																								}
																}

																int i2 = 0;
																float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																					pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

																for (int i2 = 1; i2 < 3*3; ++i2) {
																								float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																														pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																								if (dist >= max)
																																max = dist;
																}

																if( max>= maxValue2[y*width+x] ) {
																								int m = 0;
																								for(int j = 0; j < 3; ++j) {
																																for(int i = 0; i < 3; ++i) {
																																								m = j*3+i;
																																								// center
																																								if(match[(y*3+j)*width*3+(x*3+i)] == true) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][1]*backup3H[(y*3+j)*width*3+(x*3+i)]
																																																								+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][1]*backup3S[(y*3+j)*width*3+(x*3+i)]
																																																								+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][1]*backup3V[(y*3+j)*width*3+(x*3+i)] +
																																																								alpha*gaussKernel[1][1]*(frameV[y*width+x]);
																																								}
																																								// left up
																																								if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 &&
																																														match[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][2]*backup3H[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																																								+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][2]*backup3S[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																																								+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][2]*backup3V[((y+ylu[m])*3+jlu[m])*width*3+(x+xlu[m])*3+ilu[m]]
																																																								+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
																																								}
																																								// up
																																								if (  (y+yu[m])>=0 &&
																																														match[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][1]*backup3H[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																																								+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][1]*backup3S[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																																								+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][1]*backup3V[((y+yu[m])*3+ju[m])*width*3+(x+xu[m])*3+iu[m]]
																																																								+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
																																								}
																																								// right up
																																								if (  (x+xru[m])<=width && (y+yru[m])>=0 &&
																																														match[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][0]*backup3H[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																																								+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][0]*backup3S[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																																								+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[2][0]*backup3V[((y+yru[m])*3+jru[m])*width*3+(x+xru[m])*3+iru[m]]
																																																								+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
																																								}
																																								// right
																																								if (  (x+xr[m])<=width &&
																																														match[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][0]*backup3H[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																																								+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][0]*backup3S[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																																								+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][0]*backup3V[((y+yr[m])*3+jr[m])*width*3+(x+xr[m])*3+ir[m]]
																																																								+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
																																								}
																																								// right down
																																								if (  (x+xrd[m])<=width && (y+yrd[m])>=height &&
																																														match[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][0]*backup3H[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																																								+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][0]*backup3S[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																																								+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][0]*backup3V[((y+yrd[m])*3+jrd[m])*width*3+(x+xrd[m])*3+ird[m]]
																																																								+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
																																								}
																																								// down
																																								if (  (y+yd[m])>=height &&
																																														match[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][1]*backup3H[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																																								+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][1]*backup3S[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																																								+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][1]*backup3V[((y+yd[m])*3+jd[m])*width*3+(x+xd[m])*3+id[m]]
																																																								+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
																																								}
																																								// left down7
																																								if (  (y+yld[m])>=height && (x+xld[m])>=0 &&
																																														match[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][2]*backup3H[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																																								+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][2]*backup3S[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																																								+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[0][2]*backup3V[((y+yld[m])*3+jld[m])*width*3+(x+xld[m])*3+ild[m]]
																																																								+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
																																								}
																																								// left
																																								if (  (x+xl[m])>=0 &&
																																														match[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ) {
																																																model3H[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][2]*backup3H[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																																								+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
																																																model3S[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][2]*backup3S[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																																								+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
																																																model3V[(y*3+j)*width*3+(x*3+i)] =
																																																								(1-alpha)*gaussKernel[1][2]*backup3V[((y+yl[m])*3+jl[m])*width*3+(x+xl[m])*3+il[m]]
																																																								+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
																																								}
																																}
																								}
																} // if
								} // if
}

__global__ void initMean(float* modelH, float* modelS, float* modelV,
																									float* frameH, float* frameS, float* frameV,
																									float* meanDistance,
																									int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								meanDistance[(y*3+j)*width*3+(x*3+i)] =
																																distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																									modelH[(y*3+j)*width*3+(x*3+i)],
																																									modelS[(y*3+j)*width*3+(x*3+i)],
																																									modelV[(y*3+j)*width*3+(x*3+i)]);
																}
								}
}

__global__ void meanSum(float* modelH, float* modelS, float* modelV,
																								float* frameH, float* frameS, float* frameV,
																								float* meanDistance,
																								int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								meanDistance[(y*3+j)*width*3+(x*3+i)] = (
																																meanDistance[(y*3+j)*width*3+(x*3+i)] +
																																distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																									modelH[(y*3+j)*width*3+(x*3+i)],
																																									modelS[(y*3+j)*width*3+(x*3+i)],
																																									modelV[(y*3+j)*width*3+(x*3+i)])
																																)/2;
																}
								}
}

__global__ void meanSumWithFilter(float* model1H, float* model1S, float* model1V,
																																		float* model2H, float* model2S, float* model2V,
																																		float* frameH, float* frameS, float* frameV,
																																		float* maxDistance,
																																		float* meanDistance,
																																		int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
																}
								}

								int i2 = 0;
								float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

								for (int i2 = 1; i2 < 3*3; ++i2) {
																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																if (dist >= max)
																								max = dist;
								}

								if( max >= maxDistance[y*width+x] ) {
																for (int j = 0; j < 3; ++j) {
																								for (int i = 0; i < 3; ++i) {
																																meanDistance[(y*3+j)*width*3+(x*3+i)] = (
																																								meanDistance[(y*3+j)*width*3+(x*3+i)] +
																																								distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																																	model2H[(y*3+j)*width*3+(x*3+i)],
																																																	model2S[(y*3+j)*width*3+(x*3+i)],
																																																	model2V[(y*3+j)*width*3+(x*3+i)])
																																								)/2;
																								}
																}
								} // if
}

__global__ void meanSumWithFilter2(float* model1H, float* model1S, float* model1V,
																																			float* model2H, float* model2S, float* model2V,
																																			float* model3H, float* model3S, float* model3V,
																																			float* frameH, float* frameS, float* frameV,
																																			float* maxDistance1,
																																			float* maxDistance2,
																																			float* meanDistance,
																																			int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// used to calculate the distance
								float pointFrame[3];
								float pointModel[9][3];

								pointFrame[0] = frameH[y*width + x];
								pointFrame[1] = frameS[y*width + x];
								pointFrame[2] = frameV[y*width + x];

								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								pointModel[j*3+i][0] = model1H[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][1] = model1S[(y*3+j)*width*3+(x*3+i)];
																								pointModel[j*3+i][2] = model1V[(y*3+j)*width*3+(x*3+i)];
																}
								}

								int i2 = 0;
								float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																													pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

								for (int i2 = 1; i2 < 3*3; ++i2) {
																float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																						pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																if (dist >= max)
																								max = dist;
								}

								if( max >= maxDistance1[y*width+x] ) {
																for (int j = 0; j < 3; ++j) {
																								for (int i = 0; i < 3; ++i) {
																																pointModel[j*3+i][0] = model2H[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][1] = model2S[(y*3+j)*width*3+(x*3+i)];
																																pointModel[j*3+i][2] = model2V[(y*3+j)*width*3+(x*3+i)];
																								}
																}

																int i2 = 0;
																float max = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																					pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);

																for (int i2 = 1; i2 < 3*3; ++i2) {
																								float dist = distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																														pointModel[i2][0], pointModel[i2][1], pointModel[i2][2]);
																								if (dist >= max)
																																max = dist;
																}

																if( max>= maxDistance2[y*width+x] ) {
																								for (int j = 0; j < 3; ++j) {
																																for (int i = 0; i < 3; ++i) {
																																								meanDistance[(y*3+j)*width*3+(x*3+i)] = (
																																																meanDistance[(y*3+j)*width*3+(x*3+i)] +
																																																distance(pointFrame[0], pointFrame[1], pointFrame[2],
																																																									model3H[(y*3+j)*width*3+(x*3+i)],
																																																									model3S[(y*3+j)*width*3+(x*3+i)],
																																																									model3V[(y*3+j)*width*3+(x*3+i)])
																																																)/2;
																																}
																								}
																} // if
								} // if
}

__global__ void detection(float* inputH, float* inputS, float* inputV,
																										float* layer1H, float* layer1S, float* layer1V,
																										float* layer2H, float* layer2S, float* layer2V,
																										float* layer3H, float* layer3S, float* layer3V,
																										float* thresholdLayer1, float* thresholdLayer2, float* thresholdLayer3,
																										float* ouput,
																										float* labelLayerMatch,
																										bool* matchLayer1, bool* matchLayer2, bool* matchLayer3,
																										int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								float min1 = distance(inputH[y*width+x], inputS[y*width+x], inputV[y*width+x],
																														layer1H[(y*3)*width*3+x*3], layer1S[(y*3)*width*3+x*3], layer1V[(y*3)*width*3+x*3]);
								int index1 = 0;
								for(int j = 0; j < 3; ++j) {
																for(int i = 0; i < 3; ++i) {
																								float distTemp = distance(inputH[y*width+x], inputS[y*width+x], inputV[y*width+x],
																																																		layer1H[(y*3+j)*width*3+(x*3+i)],
																																																		layer1S[(y*3+j)*width*3+(x*3+i)], layer1V[(y*3+j)*width*3+(x*3+i)]);
																								if(distTemp <= min1) {
																																min1 = distTemp;
																																index1 = j*3 + i;
																								}
																}
								}

								for(int j = 0; j < 3; ++j) {
																for(int i =0; i < 3; ++i) {
																								matchLayer1[(y*3+j)*width*3+(x*3+i)] = false;
																								matchLayer2[(y*3+j)*width*3+(x*3+i)] = false;
																								matchLayer3[(y*3+j)*width*3+(x*3+i)] = false;
																}
								}

								if( min1 <= thresholdLayer1[y*width+x] ) {
																ouput[y*width+x] = 0;
																labelLayerMatch[y*width+x] = 1;
																matchLayer1[(y*3+mj[index1])*width*3+(x*3+mi[index1])] = true;
								}
								else{
																float min2 = distance(inputH[y*width+x], inputS[y*width+x], inputV[y*width+x],
																																						layer2H[(y*3)*width*3+x*3], layer2S[(y*3)*width*3+x*3], layer2V[(y*3)*width*3+x*3]);
																int index2 = 0;
																for(int j = 0; j < 3; ++j) {
																								for(int i = 0; i < 3; ++i) {
																																float distTemp = distance(inputH[y*width+x], inputS[y*width+x], inputV[y*width+x],
																																																										layer2H[(y*3+j)*width*3+(x*3+i)],
																																																										layer2S[(y*3+j)*width*3+(x*3+i)], layer2V[(y*3+j)*width*3+(x*3+i)]);
																																if(distTemp <= min2) {
																																								min2 = distTemp;
																																								index2 = j*3 + i;
																																}
																								} // i
																} // j

																if( min2 <= thresholdLayer2[y*width+x] ) {
																								ouput[y*width+x] = 0;
																								labelLayerMatch[y*width+x] = 2;
																								matchLayer2[(y*3+mj[index2])*width*3+(x*3+mi[index2])] = true;
																}
																else{
																								float min3 = distance(inputH[y*width+x], inputS[y*width+x], inputV[y*width+x],
																																														layer3H[(y*3)*width*3+x*3], layer3S[(y*3)*width*3+x*3], layer3V[(y*3)*width*3+x*3]);
																								int index3 = 0;
																								for(int j = 0; j < 3; ++j) {
																																for(int i = 0; i < 3; ++i) {
																																								float distTemp = distance(inputH[y*width+x], inputS[y*width+x], inputV[y*width+x],
																																																																		layer3H[(y*3+j)*width*3+(x*3+i)],
																																																																		layer3S[(y*3+j)*width*3+(x*3+i)], layer3V[(y*3+j)*width*3+(x*3+i)]);
																																								if(distTemp <= min3) {
																																																min3 = distTemp;
																																																index3 = j*3 + i;
																																								}
																																} // i
																								} // j

																								if( min3 <= thresholdLayer3[y*width+x] ) {
																																ouput[y*width+x] = 0;
																																labelLayerMatch[y*width+x] = 3;
																																matchLayer3[(y*3+mj[index2])*width*3+(x*3+mi[index2])] = true;
																								}
																								else{
																																ouput[y*width+x] = 1;
																																labelLayerMatch[y*width+x] = 0;
																								} // else
																} // else
								} // else
}

__global__ void update(float* frameH, float* frameS, float* frameV,
																							float* model1H, float* model1S, float* model1V,
																							float* backup1H, float* backup1S, float* backup1V,
																							float* model2H, float* model2S, float* model2V,
																							float* backup2H, float* backup2S, float* backup2V,
																							float* model3H, float* model3S, float* model3V,
																							float* backup3H, float* backup3S, float* backup3V,
																							float* labelLayerMatch,
																							bool* matchLayer1, bool* matchLayer2, bool* matchLayer3,
																							int width, int height, float alpha){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								// layer1
								if(labelLayerMatch[y*width+x] == 1) {
																int m = 0;
																for(int j = 0; j < 3; ++j) {
																								for(int i = 0; i < 3; ++i) {
																																m = j*3+i;
																																// center
																																if(matchLayer1[(y*3+j)*width*3+(x*3+i)] == true) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup1H[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup1S[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup1V[(y*3+j)*width*3+(x*3+i)] +
																																																alpha*gaussKernel[1][1]*(frameV[y*width+x]);
																																}
																																// left up
																																if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 &&
																																						matchLayer1[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup1H[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup1S[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup1V[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
																																}
																																// up
																																if (  (y+yu[m])>=0 &&
																																						matchLayer1[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup1H[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup1S[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup1V[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
																																}
																																// right up
																																if (  (x+xru[m])<=width && (y+yru[m])>=0 &&
																																						matchLayer1[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup1H[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup1S[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup1V[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
																																}
																																// right
																																if (  (x+xr[m])<=width &&
																																						matchLayer1[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup1H[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup1S[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup1V[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
																																}
																																// right down
																																if (  (x+xrd[m])<=width && (y+yrd[m])>=height &&
																																						matchLayer1[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup1H[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup1S[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup1V[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
																																}
																																// down
																																if (  (y+yd[m])>=height &&
																																						matchLayer1[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup1H[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup1S[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup1V[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
																																}
																																// left down7
																																if (  (y+yld[m])>=height && (x+xld[m])>=0 &&
																																						matchLayer1[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup1H[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup1S[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup1V[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
																																}
																																// left
																																if (  (x+xl[m])>=0 &&
																																						matchLayer1[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ) {
																																								model1H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup1H[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
																																								model1S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup1S[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
																																								model1V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup1V[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
																																}
																								}
																}
								}

								// layer2
								if(labelLayerMatch[y*width+x] == 2) {
																int m = 0;
																for(int j = 0; j < 3; ++j) {
																								for(int i = 0; i < 3; ++i) {
																																m = j*3+i;
																																// center
																																if(matchLayer2[(y*3+j)*width*3+(x*3+i)] == true) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup2H[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup2S[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup2V[(y*3+j)*width*3+(x*3+i)] +
																																																alpha*gaussKernel[1][1]*(frameV[y*width+x]);
																																}
																																// left up
																																if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 &&
																																						matchLayer2[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup2H[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup2S[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup2V[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
																																}
																																// up
																																if (  (y+yu[m])>=0 &&
																																						matchLayer2[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup2H[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup2S[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup2V[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
																																}
																																// right up
																																if (  (x+xru[m])<=width && (y+yru[m])>=0 &&
																																						matchLayer2[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup2H[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup2S[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup2V[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
																																}
																																// right
																																if (  (x+xr[m])<=width &&
																																						matchLayer2[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup2H[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup2S[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup2V[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
																																}
																																// right down
																																if (  (x+xrd[m])<=width && (y+yrd[m])>=height &&
																																						matchLayer2[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup2H[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup2S[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup2V[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
																																}
																																// down
																																if (  (y+yd[m])>=height &&
																																						matchLayer2[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup2H[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup2S[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup2V[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
																																}
																																// left down7
																																if (  (y+yld[m])>=height && (x+xld[m])>=0 &&
																																						matchLayer2[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup2H[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup2S[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup2V[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
																																}
																																// left
																																if (  (x+xl[m])>=0 &&
																																						matchLayer2[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ) {
																																								model2H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup2H[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
																																								model2S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup2S[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
																																								model2V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup2V[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
																																}
																								}
																}
								}

								// layer 3
								if(labelLayerMatch[y*width+x] == 3) {
																int m = 0;
																for(int j = 0; j < 3; ++j) {
																								for(int i = 0; i < 3; ++i) {
																																m = j*3+i;
																																// center
																																if(matchLayer3[(y*3+j)*width*3+(x*3+i)] == true) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup3H[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameH[y*width+x]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup3S[(y*3+j)*width*3+(x*3+i)]
																																																+ alpha*gaussKernel[1][1]*(frameS[y*width+x]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][1])*backup3V[(y*3+j)*width*3+(x*3+i)] +
																																																alpha*gaussKernel[1][1]*(frameV[y*width+x]);
																																}
																																// left up
																																if (  (x+xlu[m])>=0 && (y+ylu[m])>=0 &&
																																						matchLayer3[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i] == true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup3H[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameH[(y+ylu[m])*width+(x+xlu[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup3S[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameS[(y+ylu[m])*width+(x+xlu[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][2])*backup3V[((y+ylu[m])*3+j)*width*3+(x+xlu[m])*3+i]
																																																+ alpha*gaussKernel[2][2]*(frameV[(y+ylu[m])*width+(x+xlu[m])]);
																																}
																																// up
																																if (  (y+yu[m])>=0 &&
																																						matchLayer3[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i] == true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup3H[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameH[(y+yu[m])*width+(x+xu[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup3S[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameS[(y+yu[m])*width+(x+xu[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][1])*backup3V[((y+yu[m])*3+j)*width*3+(x+xu[m])*3+i]
																																																+ alpha*gaussKernel[2][1]*(frameV[(y+yu[m])*width+(x+xu[m])]);
																																}
																																// right up
																																if (  (x+xru[m])<=width && (y+yru[m])>=0 &&
																																						matchLayer3[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i] == true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup3H[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[2][0]*(frameH[(y+yru[m])*width+(x+xru[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup3S[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[2][0]*(frameS[(y+yru[m])*width+(x+xru[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[2][0])*backup3V[((y+yru[m])*3+j)*width*3+(x+xru[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yru[m])*width+(x+xru[m])]);
																																}
																																// right
																																if (  (x+xr[m])<=width &&
																																						matchLayer3[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i] == true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup3H[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameH[(y+yr[m])*width+(x+xr[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup3S[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameS[(y+yr[m])*width+(x+xr[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][0])*backup3V[((y+yr[m])*3+j)*width*3+(x+xr[m])*3+i]
																																																+ alpha*gaussKernel[1][0]*(frameV[(y+yr[m])*width+(x+xr[m])]);
																																}
																																// right down
																																if (  (x+xrd[m])<=width && (y+yrd[m])>=height &&
																																						matchLayer3[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i] ==true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup3H[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameH[(y+yrd[m])*width+(x+xrd[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup3S[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameS[(y+yrd[m])*width+(x+xrd[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][0])*backup3V[((y+yrd[m])*3+j)*width*3+(x+xrd[m])*3+i]
																																																+ alpha*gaussKernel[0][0]*(frameV[(y+yrd[m])*width+(x+xrd[m])]);
																																}
																																// down
																																if (  (y+yd[m])>=height &&
																																						matchLayer3[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i] == true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup3H[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameH[(y+yd[m])*width+(x+xd[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup3S[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameS[(y+yd[m])*width+(x+xd[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][1])*backup3V[((y+yd[m])*3+j)*width*3+(x+xd[m])*3+i]
																																																+ alpha*gaussKernel[0][1]*(frameV[(y+yd[m])*width+(x+xd[m])]);
																																}
																																// left down7
																																if (  (y+yld[m])>=height && (x+xld[m])>=0 &&
																																						matchLayer3[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i] == true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup3H[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameH[(y+yld[m])*width+(x+xld[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup3S[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameS[(y+yld[m])*width+(x+xld[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[0][2])*backup3V[((y+yld[m])*3+j)*width*3+(x+xld[m])*3+i]
																																																+ alpha*gaussKernel[0][2]*(frameV[(y+yld[m])*width+(x+xld[m])]);
																																}
																																// left
																																if (  (x+xl[m])>=0 &&
																																						matchLayer3[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i] == true  ) {
																																								model3H[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup3H[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameH[(y+yl[m])*width+(x+xl[m])]);
																																								model3S[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup3S[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameS[(y+yl[m])*width+(x+xl[m])]);
																																								model3V[(y*3+j)*width*3+(x*3+i)] =
																																																(1-alpha*gaussKernel[1][2])*backup3V[((y+yl[m])*3+j)*width*3+(x+xl[m])*3+i]
																																																+ alpha*gaussKernel[1][2]*(frameV[(y+yl[m])*width+(x+xl[m])]);
																																}
																								}
																}
								}
}

// with training
__global__ void calculateThreshold(float* meanValue, float* maxValue,
																																			int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								float tempMax = meanValue[(y*3)*width*3+(x*3)];
								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								if( meanValue[(y*3+j)*width*3+(x*3+i)]>=tempMax )
																																tempMax = meanValue[(y*3+j)*width*3+(x*3+i)];
																}
								}
								maxValue[y*width+x] = tempMax;
}

// without training, we set the minimum value of tao as 0.06
__global__ void calculateThresholdWithoutTraining(float* meanValue, float* maxValue,
																																																		int width){
								int x = blockDim.x*blockIdx.x + threadIdx.x;
								int y = blockDim.y*blockIdx.y + threadIdx.y;

								float tempMax = meanValue[(y*3)*width*3+(x*3)];
								for (int j = 0; j < 3; ++j) {
																for (int i = 0; i < 3; ++i) {
																								if( meanValue[(y*3+j)*width*3+(x*3+i)]>=tempMax )
																																tempMax = meanValue[(y*3+j)*width*3+(x*3+i)];
																}
								}
								maxValue[y*width+x] = tempMax;

								// can be adjusted by hand
								if( maxValue[y*width+x]<= 0.06 )
																maxValue[y*width+x] = 0.06;
}

void help(){
								cout<<">----------------------------------------------------------------------------------------------------------------<"<<endl<<endl;
								cout<<"The command format is:"<<endl<<endl;
								cout<<"1. smsom train <start_frame_number> <end_frame_number> <input_file_name> <output_file_name>"<<endl<<endl;
								cout<<"2. smsom train <start_frame_number> <end_frame_number> <input_file_name>"<<endl<<endl;
								cout<<"3. smsom nottrain <input_file_name> <output_file_name>"<<endl<<endl;
								cout<<"4. smsom nottrain <input_file_name> <output_file_name>"<<endl<<endl;
								cout<<"Please see http://zhaozj89.github.io/SMSOM/ for more details"<<endl<<endl;
								cout<<"Press 'q' to exit"<<endl<<endl;
								cout<<">----------------------------------------------------------------------------------------------------------------<"<<endl;
}

//
float c1 = 1;
float c2 = 0.03;
float alphaLearning = c1*4; // c1/max weight of the Gaussian kernel
float alphaAdaption = c2*4; // c2/max weight of the Gaussian kernel
int startFrame, endFrame;
int initFrame = 1;

//
bool IsTraining;
bool IsOuput;
char fileName[200];
char outputFileName[200];
char path[200];
char outputPath[200];

int main(int argv, char* argc[]){

								//
								if(argv < 3) {
																help();
																return 0;
								}

								//
								string p2(argc[1]);
								string tempTrain = "train";
								string tempNottrain = "nottrain";
								if(p2 == tempTrain) IsTraining = true;
								else if(p2 == tempNottrain) IsTraining = false;
								else{
																help();
																return 0;
								}

								//
								if(IsTraining == true) {
																if(argv == 6) {
																								startFrame = atoi(argc[2]);
																								endFrame = atoi(argc[3]);
																								strcpy(path, argc[4]);
																								strcpy(outputPath, argc[5]);
																								IsOuput = true;
																}
																else if(argv == 5) {
																								startFrame = atoi(argc[2]);
																								endFrame = atoi(argc[3]);
																								strcpy(path, argc[4]);
																								IsOuput = false;
																}
																else{
																								help();
																								return 0;
																}
								}
								else{
																if(argv == 4) {
																								strcpy(path, argc[2]);
																								strcpy(outputPath, argc[3]);
																								IsOuput = true;
																}
																else if(argv == 3) {
																								strcpy(path, argc[2]);
																								IsOuput = false;
																}
																else{
																								help();
																								return 0;
																}
								}

								// test whether input is legal
								{
																if(startFrame > endFrame) {
																								cout<<"<start_frame_number> or <end_frame_number> is  illegal, please retry!"<<endl;
																								return 0;
																}
																Mat frame;
																sprintf(fileName, path, initFrame); // read the first frame
																frame = imread(fileName, CV_LOAD_IMAGE_COLOR);
																if(frame.empty()) {
																								cout<<"<input_file_name> is illegal, please retry!"<<endl;
																								return 0;
																}
								}

								Mat frame;
								sprintf(fileName, path, initFrame); // read the first frame
								frame = imread(fileName, CV_LOAD_IMAGE_COLOR);
								int width = frame.cols;
								int height = frame.rows;

								Mat frameFloat;
								Mat frameFloat2;
								frameFloat.create(height, width, CV_32FC3);
								frameFloat2.create(height, width, CV_32FC3);
								frame.convertTo(frameFloat, CV_32FC3);
								frameFloat *= 1./255;
								cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);

								vector<Mat> input(3);
								input[0].create(height, width, CV_32FC1);
								input[1].create(height, width, CV_32FC1);
								input[2].create(height, width, CV_32FC1);
								split(frameFloat2, input);

								vector<float*> gpuInput(3);
								vector<float*> gpuLayer1(3);
								vector<float*> gpuLayer1Backup(3);
								bool* gpuMatch1;
								float* gpuOutput;
								float* gpuOutputBackup;

								Mat output;
								Mat outputFile;
								output.create(height, width, CV_32FC1);
								outputFile.create(height, width, CV_8UC3);

								for(int i = 0; i < 3; ++i) {
																hipMalloc((void**)&gpuInput[i], width*height*sizeof(float));
																hipMalloc((void**)&gpuLayer1[i], width*height*3*3*sizeof(float));
																hipMalloc((void**)&gpuLayer1Backup[i], width*height*3*3*sizeof(float));
																hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
								}
								hipMalloc((void**)&gpuMatch1, width*height*3*3*sizeof(bool));
								hipMalloc((void**)&gpuOutput, width*height*sizeof(float));
								hipMalloc((void**)&gpuOutputBackup, width*height*sizeof(float));

								dim3 grid( (width-1)/16+1, (height-1)/16+1, 1 );
								dim3 block(16, 16, 1);

								// Stacked Multi-layer Self Organizing Map Background Model (in this code, 3 layers)
								// A layer is composed of 2 parts: train and log

								// initialize layer 1
								for(int i = 0; i < 3; ++i)
																initLayer<<<grid, block>>>(gpuInput[i], gpuLayer1[i], width);

								// train layer 1
								cout<<"start training layer 1 ... ..."<<endl;
								for(int i = startFrame; i <= endFrame; ++i) {
																if(i%100 == 0)
																								cout<<"processing the "<<i<<"th image ... ..."<<endl;
																sprintf(fileName, path, i);
																frame = imread(fileName);
																if(frame.empty()) {
																								cout<<"There are no images"<<endl;
																								return 0;
																}
																frame.convertTo(frameFloat, CV_32FC3);
																frameFloat *= 1./255;
																cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
																split(frameFloat2, input);

																for(int j = 0; j < 3; ++j) {
																								hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
																								hipMemcpy(gpuLayer1Backup[j], gpuLayer1[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
																}

																compete<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																									gpuInput[0], gpuInput[1], gpuInput[2],
																																									gpuMatch1, width);
																cooperate<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																											gpuLayer1Backup[0], gpuLayer1Backup[1], gpuLayer1Backup[2],
																																											gpuInput[0], gpuInput[1], gpuInput[2],
																																											gpuMatch1,
																																											width, height, alphaLearning);
								}

								// log layer 1
								float* gpuMeanDistance1;
								float* gpuMaxDistance1;
								hipMalloc((void**)&gpuMeanDistance1, width*height*3*3*sizeof(float));
								hipMalloc((void**)&gpuMaxDistance1, width*height*sizeof(float));

								// first frame
								sprintf(fileName, path, initFrame);
								frame = imread(fileName);
								frame.convertTo(frameFloat, CV_32FC3);
								frameFloat *= 1./255;
								cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
								split(frameFloat2, input);
								for(int i = 0; i < 3; ++i)
																hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
								initMean<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																		gpuInput[0], gpuInput[1], gpuInput[2],
																																		gpuMeanDistance1, width);

								cout<<"calculate the thresholds for detection and layer 2 input ... ..."<<endl;
								for(int i = startFrame; i <= endFrame; ++i) {
																if(i%100 == 0)
																								cout<<"processing the "<<i<<"th image ... ..."<<endl;
																sprintf(fileName, path, i);
																frame = imread(fileName);
																if(frame.empty()) {
																								cout<<"There are no images"<<endl;
																								return 0;
																}
																frame.convertTo(frameFloat, CV_32FC3);
																frameFloat *= 1./255;
																cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
																split(frameFloat2, input);

																for(int j = 0; j < 3; ++j) {
																								hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
																}

																meanSum<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																									gpuInput[0], gpuInput[1], gpuInput[2],
																																									gpuMeanDistance1, width);
								}

								//
								if(IsTraining == true) {
																calculateThreshold<<<grid, block>>>(gpuMeanDistance1,
																																																				gpuMaxDistance1, width);
								}
								else{
																calculateThresholdWithoutTraining<<<grid, block>>>(gpuMeanDistance1,
																																																																			gpuMaxDistance1, width);
								}


								// train layer 2
								vector<float*> gpuLayer2(3);
								vector<float*> gpuLayer2Backup(3);
								bool* gpuMatch2;
								for (int i = 0; i < 3; ++i) {
																hipMalloc((void**)&gpuLayer2[i], width*height*3*3*sizeof(float));
																hipMalloc((void**)&gpuLayer2Backup[i], width*height*3*3*sizeof(float));
								}
								hipMalloc((void**)&gpuMatch2, width*height*3*3*sizeof(bool));

								// first frame
								sprintf(fileName, path, initFrame);
								frame = imread(fileName);
								frame.convertTo(frameFloat, CV_32FC3);
								frameFloat *= 1./255;
								cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
								split(frameFloat2, input);
								for(int i = 0; i < 3; ++i)
																hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
								for(int i = 0; i < 3; ++i)
																initLayer<<<grid, block>>>(gpuInput[i], gpuLayer2[i], width); // TODO: better initialization

								cout<<"start training layer 2 ... ..."<<endl;
								for(int i = startFrame; i <= endFrame; ++i) {
																if(i%100 == 0)
																								cout<<"processing the "<<i<<"th image ... ..."<<endl;
																sprintf(fileName, path, i);
																frame = imread(fileName);
																if(frame.empty()) {
																								cout<<"There are no images"<<endl;
																								return 0;
																}
																frame.convertTo(frameFloat, CV_32FC3);
																frameFloat *= 1./255;
																cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
																split(frameFloat2, input);

																for(int j = 0; j < 3; ++j) {
																								hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
																								hipMemcpy(gpuLayer2Backup[j], gpuLayer2[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
																}

																competeWithFilter<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																																			gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																																			gpuInput[0], gpuInput[1], gpuInput[2],
																																																			gpuMaxDistance1,
																																																			gpuMatch2, width);
																cooperateWithFilter<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																																					gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																																					gpuLayer2Backup[0], gpuLayer2Backup[1], gpuLayer2Backup[2],
																																																					gpuInput[0], gpuInput[1], gpuInput[2],
																																																					gpuMaxDistance1,
																																																					gpuMatch2,
																																																					width, height, alphaLearning);
								}

								// log layer 2
								float* gpuMeanDistance2;
								float* gpuMaxDistance2;
								hipMalloc((void**)&gpuMeanDistance2, width*height*3*3*sizeof(float));
								hipMalloc((void**)&gpuMaxDistance2, width*height*sizeof(float));

								// first frame
								sprintf(fileName, path, initFrame);
								frame = imread(fileName);
								frame.convertTo(frameFloat, CV_32FC3);
								frameFloat *= 1./255;
								cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
								split(frameFloat2, input);
								for(int i = 0; i < 3; ++i)
																hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
								initMean<<<grid, block>>>(gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																		gpuInput[0], gpuInput[1], gpuInput[2],
																																		gpuMeanDistance2, width);

								cout<<"calculate the thresholds for detection and layer 3 input ... ..."<<endl;
								for(int i = startFrame; i <= endFrame; ++i) {
																if(i%100 == 0)
																								cout<<"processing the "<<i<<"th image ... ..."<<endl;
																sprintf(fileName, path, i);
																frame = imread(fileName);
																if(frame.empty()) {
																								cout<<"There are no images"<<endl;
																								return 0;
																}
																frame.convertTo(frameFloat, CV_32FC3);
																frameFloat *= 1./255;
																cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
																split(frameFloat2, input);

																for(int j = 0; j < 3; ++j) {
																								hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
																}

																meanSumWithFilter<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																																			gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																																			gpuInput[0], gpuInput[1], gpuInput[2],
																																																			gpuMaxDistance1,
																																																			gpuMeanDistance2, width);
								}

								//
								if(IsTraining == true) {
																calculateThreshold<<<grid, block>>>(gpuMeanDistance1,
																																																				gpuMaxDistance1, width);
								}
								else{
																calculateThresholdWithoutTraining<<<grid, block>>>(gpuMeanDistance1,
																																																																			gpuMaxDistance1, width);
								}

								// train layer 3
								vector<float*> gpuLayer3(3);
								vector<float*> gpuLayer3Backup(3);
								bool* gpuMatch3;
								for (int i = 0; i < 3; ++i) {
																hipMalloc((void**)&gpuLayer3[i], width*height*3*3*sizeof(float));
																hipMalloc((void**)&gpuLayer3Backup[i], width*height*3*3*sizeof(float));
								}
								hipMalloc((void**)&gpuMatch3, width*height*3*3*sizeof(bool));

								// first frame
								sprintf(fileName, path, initFrame);
								frame = imread(fileName);
								frame.convertTo(frameFloat, CV_32FC3);
								frameFloat *= 1./255;
								cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
								split(frameFloat2, input);
								for(int i = 0; i < 3; ++i)
																hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
								for(int i = 0; i < 3; ++i)
																initLayer<<<grid, block>>>(gpuInput[i], gpuLayer3[i], width); // TODO: better initialization

								cout<<"start training layer 3 ... ..."<<endl;
								for(int i = startFrame; i <= endFrame; ++i) {
																if(i%100 == 0)
																								cout<<"processing the "<<i<<"th image ... ..."<<endl;
																sprintf(fileName, path, i);
																frame = imread(fileName);
																if(frame.empty()) {
																								cout<<"There are no images"<<endl;
																								return 0;
																}
																frame.convertTo(frameFloat, CV_32FC3);
																frameFloat *= 1./255;
																cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
																split(frameFloat2, input);

																for(int j = 0; j < 3; ++j) {
																								hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
																								hipMemcpy(gpuLayer3Backup[j], gpuLayer3[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
																}

																competeWithFilter2<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																																				gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																																				gpuLayer3[0], gpuLayer3[1], gpuLayer3[2],
																																																				gpuInput[0], gpuInput[1], gpuInput[2],
																																																				gpuMaxDistance1,
																																																				gpuMaxDistance2,
																																																				gpuMatch3, width);
																cooperateWithFilter2<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																																						gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																																						gpuLayer3[0], gpuLayer3[1], gpuLayer3[2],
																																																						gpuLayer3Backup[0], gpuLayer3Backup[1], gpuLayer3Backup[2],
																																																						gpuInput[0], gpuInput[1], gpuInput[2],
																																																						gpuMaxDistance1,
																																																						gpuMaxDistance2,
																																																						gpuMatch3,
																																																						width, height, alphaLearning);
								}

								// log layer 3
								float* gpuMeanDistance3;
								float* gpuMaxDistance3;
								hipMalloc((void**)&gpuMeanDistance3, width*height*3*3*sizeof(float));
								hipMalloc((void**)&gpuMaxDistance3, width*height*sizeof(float));

								// first frame
								sprintf(fileName, path, initFrame);
								frame = imread(fileName);
								frame.convertTo(frameFloat, CV_32FC3);
								frameFloat *= 1./255;
								cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
								split(frameFloat2, input);
								for(int i = 0; i < 3; ++i)
																hipMemcpy(gpuInput[i], input[i].data, width*height*sizeof(float), hipMemcpyHostToDevice);
								initMean<<<grid, block>>>(gpuLayer3[0], gpuLayer3[1], gpuLayer3[2],
																																		gpuInput[0], gpuInput[1], gpuInput[2],
																																		gpuMeanDistance3, width);

								cout<<"calculate the thresholds for detection and layer 4 input ... ..."<<endl;
								for(int i = startFrame; i <= endFrame; ++i) {
																if(i%100 == 0)
																								cout<<"processing the "<<i<<"th image ... ..."<<endl;
																sprintf(fileName, path, i);
																frame = imread(fileName);
																if(frame.empty()) {
																								cout<<"There are no images"<<endl;
																								return 0;
																}
																frame.convertTo(frameFloat, CV_32FC3);
																frameFloat *= 1./255;
																cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
																split(frameFloat2, input);

																for(int j = 0; j < 3; ++j) {
																								hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
																}

																meanSumWithFilter2<<<grid, block>>>(gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																																				gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																																				gpuLayer3[0], gpuLayer3[1], gpuLayer3[2],
																																																				gpuInput[0], gpuInput[1], gpuInput[2],
																																																				gpuMaxDistance1,
																																																				gpuMaxDistance2,
																																																				gpuMeanDistance3, width);
								}

								//
								if(IsTraining == true) {
																calculateThreshold<<<grid, block>>>(gpuMeanDistance1,
																																																				gpuMaxDistance1, width);
								}
								else{
																calculateThresholdWithoutTraining<<<grid, block>>>(gpuMeanDistance1,
																																																																			gpuMaxDistance1, width);
								}

								//////////////////////////////////////////////////////////////////////////
								////DEBUG
								//vector<Mat> tempLayer(3);
								//char tempName[200];
								//for(int i = 0; i < 3; ++i){
								//	tempLayer[i].create(height*3, width*3, CV_32FC1);
								//	hipMemcpy(tempLayer[i].data, gpuLayer1[i], width*height*3*3*sizeof(float), hipMemcpyDeviceToHost);
								//	namedWindow("temp", 1);
								//	imshow("temp", tempLayer[i]);
								//	waitKey(0);
								//	sprintf(tempName, "layer1_%d.png", i);
								//	imwrite(tempName, tempLayer[i]);
								//}

								//for(int i = 0; i < 3; ++i){
								//	tempLayer[i].create(height*3, width*3, CV_32FC1);
								//	hipMemcpy(tempLayer[i].data, gpuLayer2[i], width*height*3*3*sizeof(float), hipMemcpyDeviceToHost);
								//	namedWindow("temp", 1);
								//	imshow("temp", tempLayer[i]);
								//	waitKey(0);
								//	sprintf(tempName, "layer2_%d.png", i);
								//	imwrite(tempName, tempLayer[i]);
								//}
								//for(int i = 0; i < 3; ++i){
								//	tempLayer[i].create(height*3, width*3, CV_32FC1);
								//	hipMemcpy(tempLayer[i].data, gpuLayer3[i], width*height*3*3*sizeof(float), hipMemcpyDeviceToHost);
								//	namedWindow("temp", 1);
								//	imshow("temp", tempLayer[i]);
								//	waitKey(0);
								//	sprintf(tempName, "layer3_%d.png", i);
								//	imwrite(tempName, tempLayer[i]);
								//}

								//vector<Mat> tempMax(3);
								//for(int i = 0; i < 3; ++i){
								//	tempMax[i].create(height, width, CV_32FC1);
								//}
								//hipMemcpy(tempMax[0].data, gpuMaxDistance1, height*width*sizeof(float), hipMemcpyDeviceToHost);
								//hipMemcpy(tempMax[1].data, gpuMaxDistance2, height*width*sizeof(float), hipMemcpyDeviceToHost);
								//hipMemcpy(tempMax[2].data, gpuMaxDistance3, height*width*sizeof(float), hipMemcpyDeviceToHost);
								//char tempName[200];
								//vector<Mat> tempMaxInt(3);
								//for (int i = 0; i < 3; ++i){
								//	namedWindow("temp", 1);
								//	imshow("temp", tempMax[i]);
								//	waitKey(0);
								//	//tempMaxInt[i].create(height, width, CV_8UC1);
								//	//tempMax[i] *= 255;
								//	//tempMax[i].convertTo(tempMaxInt[i], CV_8UC1);
								//	//sprintf(tempName, "max_%d.png", i);
								//	//imwrite(tempName, tempMaxInt[i]);
								//}

								//for(int k = 0; k < 3; ++k){
								//	float* p;
								//	float maxV = -10.0;
								//	for(int i = 0; i < height; ++i){
								//		p = tempMax[k].ptr<float>(i);
								//		for(int j = 0; j < width; ++j){
								//			if( p[j] >= maxV )
								//				maxV = p[j];
								//		}
								//	}

								//	cout<<maxV<<endl;
								//}

								//return 0;
								//////////////////////////////////////////////////////////////////////////


								// start detection and update SMSOM on-line
								float* gpuLabelLayerMatch;
								hipMalloc((void**)&gpuLabelLayerMatch, width*height*sizeof(float));
								cout<<"start detecting the foreground on-line ... ..."<<endl;
								char key = NULL;
								int frameNum = endFrame + 1;
								// clock_t startTime = clock();
								namedWindow("foreground", 1);
								// for(frameNum = 165; frameNum <= 300; ++frameNum){
								while (key != 'q') {
																++frameNum;
																if(frameNum%100 == 0)
																								cout<<"processing the "<<frameNum<<"th image ... ..."<<endl;
																sprintf(fileName, path, frameNum);
																frame = imread(fileName);
																if(frame.empty()) {
																								cout<<"There are no images"<<endl;
																								return 0;
																}
																frame.convertTo(frameFloat, CV_32FC3);
																frameFloat *= 1./255;
																cvtColor(frameFloat, frameFloat2, CV_BGR2HSV);
																split(frameFloat2, input);

																for(int j = 0; j < 3; ++j) {
																								hipMemcpy(gpuInput[j], input[j].data, width*height*sizeof(float), hipMemcpyHostToDevice);
																								hipMemcpy(gpuLayer1Backup[j], gpuLayer1[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
																								hipMemcpy(gpuLayer2Backup[j], gpuLayer2[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
																								hipMemcpy(gpuLayer3Backup[j], gpuLayer3[j], width*height*3*3*sizeof(float), hipMemcpyDeviceToDevice);
																}

																detection<<<grid, block>>>(gpuInput[0], gpuInput[1], gpuInput[2],
																																											gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																											gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																											gpuLayer3[0], gpuLayer3[1], gpuLayer3[2],
																																											gpuMaxDistance1, gpuMaxDistance2, gpuMaxDistance3,
																																											gpuOutput,
																																											gpuLabelLayerMatch,
																																											gpuMatch1, gpuMatch2, gpuMatch3,
																																											width);
																update<<<grid, block>>>(gpuInput[0], gpuInput[1], gpuInput[2],
																																								gpuLayer1[0], gpuLayer1[1], gpuLayer1[2],
																																								gpuLayer1Backup[0], gpuLayer1Backup[1], gpuLayer1Backup[2],
																																								gpuLayer2[0], gpuLayer2[1], gpuLayer2[2],
																																								gpuLayer2Backup[0], gpuLayer2Backup[1], gpuLayer2Backup[2],
																																								gpuLayer3[0], gpuLayer3[1], gpuLayer3[2],
																																								gpuLayer3Backup[0], gpuLayer3Backup[1], gpuLayer3Backup[2],
																																								gpuLabelLayerMatch,
																																								gpuMatch1, gpuMatch2, gpuMatch3,
																																								width, height, alphaAdaption
																																								);

																//Mat tempLabel;
																//tempLabel.create(height, width, CV_32FC1);
																//hipMemcpy(tempLabel.data, gpuLabelLayerMatch, width*height*sizeof(float), hipMemcpyDeviceToHost);
																//tempLabel /= 3.0;
																//namedWindow("temp", 1);
																//imshow("temp", tempLabel);
																//waitKey(0);

																hipMemcpy(output.data, gpuOutput, width*height*sizeof(float), hipMemcpyDeviceToHost);
																imshow("foreground", output);
																if(IsOuput == true) {
																								sprintf(outputFileName, outputPath, frameNum);
																								output *= 255;
																								output.convertTo(outputFile, CV_8UC3);
																								imwrite(outputFileName, outputFile);
																}
																key = waitKey(1);
								}
								//clock_t endTime = clock();
								//cout<<"startTime="<<startTime<<endl;
								//cout<<"endTime="<<endTime<<endl;
								//cout<<"speed="<<(double)(endTime-startTime)/CLOCKS_PER_SEC<<endl;
								//DEBUG
								//for(int i = 0; i < 3; ++i){
								//	Mat outputTemp;
								//	outputTemp.create(height*3, width*3, CV_32FC1);
								//	hipMemcpy(outputTemp.data, gpuLayer2[i], width*height*3*3*sizeof(float), hipMemcpyDeviceToHost);
								//	namedWindow("layer2", 1);
								//	imshow("layer2", outputTemp/360);
								//	waitKey(0);
								//}


								//DEBUG
								//hipMemcpy(output.data, gpuMaxDistance1, width*height*sizeof(float), hipMemcpyDeviceToHost);
								//namedWindow("max", 1);
								//imshow("max", output);
								//waitKey(0);

								//hipMemcpy(output.data, gpuThreshold1, width*height*sizeof(float), hipMemcpyDeviceToHost);
								//namedWindow("threshold", 1);
								//imshow("threshold", output);
								//waitKey(0);
								return 0;
}
